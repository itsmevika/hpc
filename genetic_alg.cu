#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"	 
#include ""
#include <iostream>
#include <stdio.h>
#include <algorithm>
#include <ctime>
#include <cstdlib> 

using namespace std;

const int NumberOfPoint = 500; //количество точек
const int NumberOfIndividov = 1000; //кол-во индивидов в выборке
const int MathMutation = 5; //мутации 
const double dispersionMutation = 5.0f; //максимальная мутация
const int powCount = 3;
const double randMaxCount = 20.0f; //максимальный разброс рандома
const int KolOfPokoleni = 30; //максимальное кол-во поколений


__global__ void Errors(double* points, double* individs, double* errors, int powCount, int NumberOfPoint) //проверяем поколение
{
	int id = threadIdx.x; //разделяем на поотоки
	double ans = 0; //ответ
	int x = 1;
	for (int i = 0; i < NumberOfPoint; i++) //перебираем точки
	{
		for (int j = 0; j < powCount; j++)
		{
			for (int k = 0; k < j; k++)
			    x *= i;

			x *= individs[id * powCount + j]; //считаем поколение
			ans += x;
			x = 1;
		}
		ans = points[i] - ans; //считаем ошибку
		errors[id] += sqrt(ans * ans);
		ans = 0;
	}
}




void testErrors(double* points, double* individs, double* errors, int powCount, int NumberOfPoint, int Random) //тоже самое что и верхняя функция, проверяем через нее индивидов
{
	for (int id = 0; id < NumberOfIndividov; id++)
	{
		double ans = 0.0f;
		errors[id] = 0.0f;
		int x = 0;
		for (int i = 0; i < NumberOfPoint; i++)
		{
			for (int j = 0; j < powCount; j++)
			{
				x = pow(i, j);
				x *= individs[id * powCount + j];
				ans += x;
				x = 0;
			}
			ans = points[i] - ans;
			errors[id] += sqrt(ans * ans);
			ans = 0;
		}
	}
}


double Random(double a, double b) { //рандомно заполняем первое поколение
	double random = ((double)rand()) / (double)RAND_MAX;
	double d = b - a;
	double r = random * d;
	return(a + r);
}

void cpu() 
{ //обработка на cpu 
	double* pointsH = new double[NumberOfPoint]; //заводим массив точек
	for (int i = 0; i < NumberOfPoint; i++) pointsH[i] = Random(0, 20); //заполняем массив рандомно 

	double* individumsH = new double[NumberOfIndividov * powCount]; //заводим массив на индивидов
	for (int i = 0; i < NumberOfIndividov * powCount; i++) individumsH[i] = Random(0, randMaxCount); //заполняем массив рандомно

	double* errorsH = new double[NumberOfIndividov]; //заводим массив ошибок для каждого индивида
	for (int i = 0; i < NumberOfIndividov; i++) errorsH[i] = 1000; //ставим максимальную ошибку

	unsigned int start_time = clock(); // начальное время

	for (int pokolenie = 0; pokolenie < KolOfPokoleni; pokolenie++) //цикл перебираем поколения
	{
		testErrors(pointsH, individumsH, errorsH, powCount, NumberOfPoint, NumberOfIndividov); //тестируем поколение на ошибку
		double* errorsCrossOver = new double[NumberOfIndividov]; //заводим массив ошибок

		for (size_t i = 0; i != NumberOfIndividov; ++i) errorsCrossOver[i] = errorsH[i]; //записываем ошибки в массив
		sort(errorsCrossOver, errorsCrossOver + NumberOfIndividov); //соритируем данное поколение 

		int merodianCrossOvering = NumberOfIndividov / 2; 
		double merodianErrorCrossOvering = errorsCrossOver[merodianCrossOvering]; //переписываем ошибки данного поколения 
		double* theBestInd = new double[powCount]; //лучшие индивиды в поколение

		for (size_t i = 0; i < NumberOfIndividov; i++)
		{
			if (merodianErrorCrossOvering < errorsH[i]) { //записываем новое поколение если только оно лучше старого
				for (size_t j = 0; j < powCount; j++) individumsH[i * powCount + j] = 0;
			}
			if (errorsH[i] == errorsCrossOver[0]) {
				for (int j = 0; j < powCount; j++) theBestInd[j] = individumsH[i * powCount + j]; //сохраняем  лучшие индивид
			}
		}
		printf("error = %f\n", errorsCrossOver[0]); //выводим каждый раз ошибку
		for (int i = 0; i < NumberOfIndividov * powCount; i++) //перебираем всех индивидов 
		{
			if (individumsH[i] == 0) {
				individumsH[i] = theBestInd[rand() % powCount]; //записываем в новое поколение лучшего и рандомного 
			}

			if (MathMutation > (rand() % 100 + 1)) {
				individumsH[i] += Random(-dispersionMutation, dispersionMutation);  //применяем мутацию
			}
		}
	}
	unsigned int end_time = clock(); // конечное время
	unsigned int search_time = end_time - start_time; // искомое время
	printf("search_time_cpu = %i\n", search_time); //выводим время обработки 
}


void gpu() {//обработка на gpu, в данной подпрограмме мы делаем все тоже самое что и cpu только с использованием библиотеки для работы с gpu
	double* pointsH = new double[NumberOfPoint];
	for (int i = 0; i < NumberOfPoint; i++) pointsH[i] = Random(0, 20);

	double* individumsH = new double[NumberOfIndividov * powCount];
	for (int i = 0; i < NumberOfIndividov * powCount; i++) individumsH[i] = Random(0, randMaxCount);

	double* errorsH = new double[NumberOfIndividov];
	for (int i = 0; i < NumberOfIndividov; i++) errorsH[i] = 1000;

	unsigned int start_time_gpu = clock(); // начальное время
	double* pointsD = NULL;
	double* individumsD = NULL;
	double* errorsD = NULL;


	for (int pokolenie = 0; pokolenie < KolOfPokoleni; pokolenie++)
	{
		int NumberOfIndividovBytes = NumberOfIndividov * powCount * sizeof(double);
		int NumberOfPointBytes = NumberOfPoint * sizeof(double);

		hipMalloc((void**)&pointsD, NumberOfPointBytes);
		hipMalloc((void**)&individumsD, NumberOfIndividovBytes * powCount);
		hipMalloc((void**)&errorsD, NumberOfIndividov * sizeof(double));

		hipMemcpy(pointsD, pointsH, NumberOfPointBytes, hipMemcpyHostToDevice);
		hipMemcpy(individumsD, individumsH, NumberOfIndividovBytes, hipMemcpyHostToDevice);
		hipMemcpy(errorsD, errorsH, NumberOfIndividovBytes, hipMemcpyHostToDevice);

		Errors << <1, NumberOfIndividov >> > (pointsD, individumsD, errorsD, powCount, NumberOfPoint);

		hipMemcpy(errorsH, errorsD, NumberOfIndividov * sizeof(double), hipMemcpyDeviceToHost);
        double* errorsCrossOver = new double[NumberOfIndividov];

		for (size_t i = 0; i != NumberOfIndividov; ++i) errorsCrossOver[i] = errorsH[i];
		
		sort(errorsCrossOver, errorsCrossOver + NumberOfIndividov);
		printf("error = %f\n", errorsCrossOver[0]);
		
		int merodianCrossOvering = NumberOfIndividov / 2;
		double merodianErrorCrossOvering = errorsCrossOver[merodianCrossOvering];
		double* theBestInd = new double[powCount];

		for (size_t i = 0; i < NumberOfIndividov; i++)
		{
			if (merodianErrorCrossOvering < errorsH[i]) {
				for (size_t j = 0; j < powCount; j++)
				{
					individumsH[i * powCount + j] = 0;
				}
			}
			if (errorsH[i] == errorsCrossOver[0]) {
				for (int j = 0; j < powCount; j++)
				{
					theBestInd[j] = individumsH[i * powCount + j];
				}
			}
		}
		for (int i = 0; i < NumberOfIndividov * powCount; i++)
		{
			if (individumsH[i] == 0) {
				individumsH[i] = theBestInd[rand() % powCount];
			}
			if (MathMutation > (rand() % 100 + 1)) {
				individumsH[i] += Random(-dispersionMutation, dispersionMutation);
			}
		}
	}
	unsigned int all_time_gpu = clock(); // все время
	unsigned int find_time_gpu = all_time_gpu - start_time_gpu; // искомое время
    printf("find_time_gpu = %i\n", find_time_gpu); //выводим время

	hipFree(pointsD); hipFree(individumsD); hipFree(errorsD); //завершаем куду

	delete pointsH;	delete individumsH; delete errorsH; //чистим память
}


int main()
{
	cpu(); //обработка на cpu
	gpu(); //обработка на gpu
	system("pause");
	return 0;
}
